#include "hip/hip_runtime.h"
#include <iostream>
#include "opencv2/imgproc.hpp"
#include "opencv2/imgcodecs.hpp"
#include "opencv2/highgui.hpp"
#include <fstream>
#include <sstream>
#include <stdlib.h>
#include "input.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <typeinfo>

using namespace std;
using namespace cv;
using namespace chrono;

Mat img_hsv, img, new_h, new_s, new_v, dst_h, dst_s, dst_v;
int total_threads;
int ksize;
vector<pair<int, int>> delta;

__global__ void median_filter_thread(const uchar *inputImageKernel,
                                     uchar *outputImagekernel,
                                     const int imageWidth,
                                     const int imageHeight) {
  // Set row and colum for thread.
  int WINDOW_SIZE = 3;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned char filterVector[9] = {0, 0, 0, 0, 0,
                                   0, 0, 0, 0}; // Take fiter window

  if ((row == 0) || (col == 0) || (row == imageHeight - 1) ||
      (col == imageWidth - 1)) {
    outputImagekernel[row * imageWidth + col] =
        0; // Deal with boundry conditions */
  } else {
    for (int x = 0; x < WINDOW_SIZE; x++) {
      for (int y = 0; y < WINDOW_SIZE; y++) {
        filterVector[x * WINDOW_SIZE + y] =
            inputImageKernel[(row + x - 1) * imageWidth +
                             (col + y - 1)]; // setup the filterign* window. */
      }
    }
    for (int i = 0; i < 9; i++) {
      for (int j = i + 1; j < 9; j++) {
        if (filterVector[i] > filterVector[j]) {
          // Swap the variables.
          char tmp = filterVector[i];
          filterVector[i] = filterVector[j];
          filterVector[j] = tmp;
        }
      }
    }
    /* outputImagekernel[row * imageWidth + col] = */
    /*     filterVector[4]; // Set the* output variables. */
    outputImagekernel[row * imageWidth + col] =
        inputImageKernel[row * imageWidth + col];
  }
}

int main(int argc, char *argv[]) {

  // Getting Arguments from execution command
  checkNumArgs(5, argc, argv[0]);

  // Parsing Arguments
  ksize = parsePosInt(argv[1]);
  total_threads = parsePosInt(argv[2]);
  img = getImg(argv[3]);

  cvtColor(img, img_hsv, COLOR_BGR2HSV);
  vector<Mat> hsvChannels(3);
  split(img_hsv, hsvChannels);

  Mat new_h = hsvChannels[0];
  Mat new_s = hsvChannels[1];
  Mat new_v = hsvChannels[2];

  hipEvent_t start_cu, stop_cu;
  hipEventCreate(&start_cu);
  hipEventCreate(&stop_cu);
  hipEventRecord(start_cu);

  hipError_t err = hipSuccess;

  // flatten the mat.
  uint totalElements = img.total(); // Note: image.total() == rows*cols.
  Mat flat_h =
      new_h.reshape(1, totalElements); // 1xN mat of 1 channel, O(1) operation
  Mat flat_s =
      new_s.reshape(1, totalElements); // 1xN mat of 1 channel, O(1) operation
  Mat flat_v =
      new_v.reshape(1, totalElements); // 1xN mat of 1 channel, O(1) operation

  if (!new_h.isContinuous()) {
    flat_h = flat_h.clone(); // O(N),
  }
  if (!new_s.isContinuous()) {
    flat_s = flat_s.clone(); // O(N),
  }
  if (!new_v.isContinuous()) {
    flat_v = flat_v.clone(); // O(N),
  }

  // flat.data is your array pointer
  auto *h_img_or_h = flat_h.data; // usually, its uchar*
  auto *h_img_or_s = flat_s.data; // usually, its uchar*
  auto *h_img_or_v = flat_v.data; // usually, its uchar*
  // You have your array, its length is flat.total() [rows=1,
  // cols=totalElements] Converting to vector
  /* std::vector<uchar> vec(flat.data, flat.data + flat.total()); */

  /*
   *
      for (vector<uchar>::const_iterator i = vec.begin(); i != vec.end(); ++i)
          cout << *i << ' ';
  Mat restored = Mat(img.rows, img.cols, img.type(), ptr);
          */
  size_t size = flat_h.total() * sizeof(uchar);

  uchar *d_img_or_h = NULL;
  err = hipMalloc((void **)&d_img_or_h, size);

  if (err != hipSuccess) {
    cout << "Error separando espacio imagen normal en GPU "
         << hipGetErrorString(err) << endl;
    return -1;
  }

  uchar *d_img_or_s = NULL;
  err = hipMalloc((void **)&d_img_or_s, size);

  if (err != hipSuccess) {
    cout << "Error separando espacio imagen normal en GPU "
         << hipGetErrorString(err) << endl;
    return -1;
  }

  uchar *d_img_or_v = NULL;
  err = hipMalloc((void **)&d_img_or_v, size);

  if (err != hipSuccess) {
    cout << "Error separando espacio imagen normal en GPU "
         << hipGetErrorString(err) << endl;
    return -1;
  }

  uchar *d_img_mf_h = NULL;

  err = hipMalloc((void **)&d_img_mf_h, size);
  if (err != hipSuccess) {
    cout << "Error separando espacio imagen normal en GPU 2"
         << hipGetErrorString(err) << endl;
    return -1;
  }

  uchar *d_img_mf_s = NULL;

  err = hipMalloc((void **)&d_img_mf_s, size);
  if (err != hipSuccess) {
    cout << "Error separando espacio imagen normal en GPU 2"
         << hipGetErrorString(err) << endl;
    return -1;
  }

  uchar *d_img_mf_v = NULL;

  err = hipMalloc((void **)&d_img_mf_v, size);
  if (err != hipSuccess) {
    cout << "Error separando espacio imagen normal en GPU 2"
         << hipGetErrorString(err) << endl;
    return -1;
  }

  uchar *h_img_mf_h = (uchar *)malloc(size);
  uchar *h_img_mf_s = (uchar *)malloc(size);
  uchar *h_img_mf_v = (uchar *)malloc(size);

  // Start timing
  auto start = high_resolution_clock::now();

  // MemoryCopy
  // Imagen
  err = hipMemcpy(d_img_or_h, h_img_or_h, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cout << "Error copiando datos a GPU " << hipGetErrorString(err) << endl;
    return -1;
  }
  err = hipMemcpy(d_img_or_s, h_img_or_s, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cout << "Error copiando datos a GPU " << hipGetErrorString(err) << endl;
    return -1;
  }
  err = hipMemcpy(d_img_or_v, h_img_or_v, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cout << "Error copiando datos a GPU " << hipGetErrorString(err) << endl;
    return -1;
  }

  // Lanzar GPU
  int blocksPerGrid = 10;
  int num_threads = 128;

  median_filter_thread<<<blocksPerGrid, num_threads>>>(d_img_or_h, d_img_mf_h,
                                                       img.rows, img.cols);

  err = hipGetLastError();
  if (err != hipSuccess) {
    cout << "Fallo al lanzar Kernel de GPU " << hipGetErrorString(err) << endl;
    return -1;
  }
  median_filter_thread<<<blocksPerGrid, num_threads>>>(d_img_or_s, d_img_mf_s,
                                                       img.rows, img.cols);

  err = hipGetLastError();
  if (err != hipSuccess) {
    cout << "Fallo al lanzar Kernel de GPU " << hipGetErrorString(err) << endl;
    return -1;
  }
  median_filter_thread<<<blocksPerGrid, num_threads>>>(d_img_or_v, d_img_mf_v,
                                                       img.rows, img.cols);

  err = hipGetLastError();
  if (err != hipSuccess) {
    cout << "Fallo al lanzar Kernel de GPU " << hipGetErrorString(err) << endl;
    return -1;
  }
  hipDeviceSynchronize();

  err = hipMemcpy(h_img_mf_h, d_img_mf_h, size, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    cout << "Error copiando datos a GPU " << hipGetErrorString(err) << endl;
    return -1;
  }
  err = hipMemcpy(h_img_mf_s, d_img_mf_s, size, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    cout << "Error copiando datos a GPU " << hipGetErrorString(err) << endl;
    return -1;
  }

  err = hipMemcpy(h_img_mf_v, d_img_mf_v, size, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    cout << "Error copiando datos a GPU " << hipGetErrorString(err) << endl;
    return -1;
  }

  // End timing
  auto end = high_resolution_clock::now();
  /* Mat restored = Mat(img.rows, img.cols, img.type(), h_img_mf); */
  /* vector<uchar> vec(h_img_mf, h_img_mf + flat.total()); */

  /* for (vector<uchar>::const_iterator i = vec.begin(); i != vec.end(); ++i) */
  /* cout << (uchar)*i << ' '; */
  /* putImg(restored, argv[4]); */

  // Delta is a vector of the offset for the windowing, and depends of ksize
  /* for (int i = 0; i < ksize; i++) { */
  /*   for (int j = 0; j < ksize; j++) { */
  /*     delta.push_back(make_pair(i, j)); */
  /*   } */
  /* } */

  duration<double, milli> total_time = (end - start);
  cout << "time    = " << total_time.count() / 1000 << '\n';

  Mat dst_h = Mat(img.rows, img.cols, CV_8UC1, h_img_mf_h);
  Mat dst_s = Mat(img.rows, img.cols, CV_8UC1, h_img_mf_s);
  Mat dst_v = Mat(img.rows, img.cols, CV_8UC1, h_img_mf_v);

  vector<uchar> vec(h_img_mf_h, h_img_mf_h + flat_h.total());

  for (vector<uchar>::const_iterator i = vec.begin(); i != vec.end(); ++i)
    cout << (uchar)*i << ' ';
  // After applied MedianFilter for each channel, were merged due to build the
  // Image Again
  vector<Mat> channels = {dst_h, dst_s, dst_v};
  Mat merged, filtered;
  merge(channels, merged);

  // Changing HSV mode color to BGR again
  cvtColor(merged, filtered, COLOR_HSV2BGR);

  // Saving Image results
  putImg(filtered, argv[4]);
  hipEventRecord(stop_cu);
  hipEventSynchronize(stop_cu);

  hipFree(d_img_or_h);
  hipFree(d_img_or_s);
  hipFree(d_img_or_v);
  hipFree(d_img_mf_h);
  hipFree(d_img_mf_s);
  hipFree(d_img_mf_v);
  /* free(h_img_or_h); */
  /* free(h_img_or_s); */
  /* free(h_img_or_v); */
  free(h_img_mf_h);
  free(h_img_mf_s);
  free(h_img_mf_v);
}
