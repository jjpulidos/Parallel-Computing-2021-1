#include "hip/hip_runtime.h"
#include <iostream>
#include "opencv2/imgproc.hpp"
#include "opencv2/imgcodecs.hpp"
#include "opencv2/highgui.hpp"
#include <fstream>
#include <sstream>
#include <stdlib.h>
#include "input.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <typeinfo>

using namespace std;
using namespace cv;
using namespace chrono;

Mat img_hsv, img, new_h, new_s, new_v, dst_h, dst_s, dst_v;
int total_threads;
int ksize;
vector<pair<int, int>> delta;

__global__ void median_filter_thread(const uchar *inputImageKernel,
                                     uchar *outputImagekernel,
                                     const int imageWidth,
                                     const int imageHeight) {
  // Set row and colum for thread.
  int WINDOW_SIZE = 3;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned char filterVector[9] = {0, 0, 0, 0, 0, 0, 0, 0, 0};

  if ((row == 0) || (col == 0) || (row == imageHeight - 1) ||
      (col == imageWidth - 1)) {
    outputImagekernel[row * imageWidth + col] = 0;
  } else {
    for (int x = 0; x < WINDOW_SIZE; x++) {
      for (int y = 0; y < WINDOW_SIZE; y++) {
        filterVector[x * WINDOW_SIZE + y] =
            inputImageKernel[(row + x - 1) * imageWidth + (col + y - 1)];
      }
    }
    for (int i = 0; i < 9; i++) {
      for (int j = i + 1; j < 9; j++) {
        if (filterVector[i] > filterVector[j]) {
          // Swap the variables.
          char tmp = filterVector[i];
          filterVector[i] = filterVector[j];
          filterVector[j] = tmp;
        }
      }
    }
    outputImagekernel[row * imageWidth + col] = filterVector[4];

    /* outputImagekernel[row * imageWidth + col] = */
    /*     inputImageKernel[row * imageWidth + col]; */
  }
}

int main(int argc, char *argv[]) {

  // Getting Arguments from execution command
  checkNumArgs(6, argc, argv[0]);

  // Parsing Arguments
  int blocksPerGrid = parsePosInt(argv[1]);
  int num_threads_x = parsePosInt(argv[4]);
  int num_threads_y = parsePosInt(argv[5]);
  img = getImg(argv[2]);

  cvtColor(img, img_hsv, COLOR_BGR2HSV);
  vector<Mat> hsvChannels(3);
  split(img_hsv, hsvChannels);

  Mat new_h = hsvChannels[0];
  Mat new_s = hsvChannels[1];
  Mat new_v = hsvChannels[2];

  hipError_t err = hipSuccess;

  // flatten the mat.
  uint totalElements = img.total(); // Note: image.total() == rows*cols.
  Mat flat_h =
      new_h.reshape(1, totalElements); // 1xN mat of 1 channel, O(1) operation
  Mat flat_s =
      new_s.reshape(1, totalElements); // 1xN mat of 1 channel, O(1) operation
  Mat flat_v =
      new_v.reshape(1, totalElements); // 1xN mat of 1 channel, O(1) operation

  if (!new_h.isContinuous()) {
    flat_h = flat_h.clone(); // O(N),
  }
  if (!new_s.isContinuous()) {
    flat_s = flat_s.clone(); // O(N),
  }
  if (!new_v.isContinuous()) {
    flat_v = flat_v.clone(); // O(N),
  }

  // flat.data is your array pointer
  auto *h_img_or_h = flat_h.data; // usually, its uchar*
  auto *h_img_or_s = flat_s.data; // usually, its uchar*
  auto *h_img_or_v = flat_v.data; // usually, its uchar*

  size_t size = img.rows * img.cols * sizeof(char);

  uchar *d_img_or_h = NULL;
  err = hipMalloc((void **)&d_img_or_h, size);

  if (err != hipSuccess) {
    cout << "Error separando espacio imagen normal en GPU "
         << hipGetErrorString(err) << endl;
    return -1;
  }

  uchar *d_img_or_s = NULL;
  err = hipMalloc((void **)&d_img_or_s, size);

  if (err != hipSuccess) {
    cout << "Error separando espacio imagen normal en GPU "
         << hipGetErrorString(err) << endl;
    return -1;
  }

  uchar *d_img_or_v = NULL;
  err = hipMalloc((void **)&d_img_or_v, size);

  if (err != hipSuccess) {
    cout << "Error separando espacio imagen normal en GPU "
         << hipGetErrorString(err) << endl;
    return -1;
  }

  uchar *d_img_mf_h = NULL;

  err = hipMalloc((void **)&d_img_mf_h, size);
  if (err != hipSuccess) {
    cout << "Error separando espacio imagen normal en GPU 2"
         << hipGetErrorString(err) << endl;
    return -1;
  }

  uchar *d_img_mf_s = NULL;

  err = hipMalloc((void **)&d_img_mf_s, size);
  if (err != hipSuccess) {
    cout << "Error separando espacio imagen normal en GPU 2"
         << hipGetErrorString(err) << endl;
    return -1;
  }

  uchar *d_img_mf_v = NULL;

  err = hipMalloc((void **)&d_img_mf_v, size);
  if (err != hipSuccess) {
    cout << "Error separando espacio imagen normal en GPU 2"
         << hipGetErrorString(err) << endl;
    return -1;
  }

  uchar *h_img_mf_h = (uchar *)malloc(size);
  uchar *h_img_mf_s = (uchar *)malloc(size);
  uchar *h_img_mf_v = (uchar *)malloc(size);

  // MemoryCopy
  // Imagen
  err = hipMemcpy(d_img_or_h, h_img_or_h, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cout << "Error copiando datos a GPU " << hipGetErrorString(err) << endl;
    return -1;
  }
  err = hipMemcpy(d_img_or_s, h_img_or_s, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cout << "Error copiando datos a GPU " << hipGetErrorString(err) << endl;
    return -1;
  }
  err = hipMemcpy(d_img_or_v, h_img_or_v, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    cout << "Error copiando datos a GPU " << hipGetErrorString(err) << endl;
    return -1;
  }

  // Lanzar GPU

  const dim3 block(blocksPerGrid, blocksPerGrid);
  if (num_threads_x == 0 || num_threads_y == 0) {
    num_threads_x = (int)ceil((float)img.cols / (float)blocksPerGrid);
    num_threads_y = (int)ceil((float)img.rows / (float)blocksPerGrid);
  }
  const dim3 grid(num_threads_x, num_threads_y);
  /* cout << (int)ceil((float)img.cols / (float)blocksPerGrid) << endl; */
  /* cout << (int)ceil((float)img.rows / (float)blocksPerGrid) << endl; */

  // Start timing
  auto start = high_resolution_clock::now();

  median_filter_thread<<<grid, block>>>(d_img_or_h, d_img_mf_h, img.cols,
                                        img.rows);

  err = hipGetLastError();
  if (err != hipSuccess) {
    cout << "Fallo al lanzar Kernel de GPU " << hipGetErrorString(err) << endl;
    return -1;
  }
  median_filter_thread<<<grid, block>>>(d_img_or_s, d_img_mf_s, img.cols,
                                        img.rows);

  err = hipGetLastError();
  if (err != hipSuccess) {
    cout << "Fallo al lanzar Kernel de GPU " << hipGetErrorString(err) << endl;
    return -1;
  }
  median_filter_thread<<<grid, block>>>(d_img_or_v, d_img_mf_v, img.cols,
                                        img.rows);

  err = hipGetLastError();
  if (err != hipSuccess) {
    cout << "Fallo al lanzar Kernel de GPU " << hipGetErrorString(err) << endl;
    return -1;
  }
  hipDeviceSynchronize();

  // End timing
  auto end = high_resolution_clock::now();

  err = hipMemcpy(h_img_mf_h, d_img_mf_h, size, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    cout << "Error copiando datos a GPU " << hipGetErrorString(err) << endl;
    return -1;
  }
  err = hipMemcpy(h_img_mf_s, d_img_mf_s, size, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    cout << "Error copiando datos a GPU " << hipGetErrorString(err) << endl;
    return -1;
  }

  err = hipMemcpy(h_img_mf_v, d_img_mf_v, size, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    cout << "Error copiando datos a GPU " << hipGetErrorString(err) << endl;
    return -1;
  }

  duration<double, milli> total_time = (end - start);
  cout << "time    = " << total_time.count() / 1000 << '\n';

  Mat dst_h = Mat(img.rows, img.cols, CV_8UC1, h_img_mf_h);
  Mat dst_s = Mat(img.rows, img.cols, CV_8UC1, h_img_mf_s);
  Mat dst_v = Mat(img.rows, img.cols, CV_8UC1, h_img_mf_v);

  vector<uchar> vec(h_img_mf_h, h_img_mf_h + flat_h.total());

  // After applied MedianFilter for each channel, were merged due to build the
  // Image Again
  vector<Mat> channels = {dst_h, dst_s, dst_v};
  Mat merged, filtered;
  merge(channels, merged);

  // Changing HSV mode color to BGR again
  cvtColor(merged, filtered, COLOR_HSV2BGR);

  putImg(filtered, argv[3]);

  // Saving Image results
  hipFree(d_img_or_h);
  hipFree(d_img_or_s);
  hipFree(d_img_or_v);
  hipFree(d_img_mf_h);
  hipFree(d_img_mf_s);
  hipFree(d_img_mf_v);
  /* free(h_img_or_h); */
  /* free(h_img_or_s); */
  /* free(h_img_or_v); */
  free(h_img_mf_h);
  free(h_img_mf_s);
  free(h_img_mf_v);
}
